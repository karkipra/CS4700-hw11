
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

__global__ void mtxMult(float *A, float *B, float *C, int N){

    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    float temp = 0;

    if(row < N && col < N){
        // compute for each thread
        for (int i = 0; i < N; i++) {
            temp += A[row * N + i] * B[i * N + col];
        }
    }
    C[row * N + col] = temp;
}

int main(){
    // Block and Tile Size
    int N = 64;
    int T = 8;
    size_t memSize = N * N * sizeof(int);
    
    printf("Running on N = %d\n", N);
    // Allocate host memory
    float* h_A;
    float* h_B;
    float* C_CPU;
    float* C_GPU;
    h_A = (float *) malloc(memSize);
    h_B = (float *) malloc(memSize);
    C_CPU = (float *) malloc(memSize);
    C_GPU = (float *) malloc(memSize);

    // Allocate device memory
    float* d_A; 
    float* d_B;
    float* d_C;
    hipMalloc((void **) &d_A, memSize);
    hipMalloc((void **) &d_B, memSize);
    hipMalloc((void **) &d_C, memSize);

    // Initialize matrices on host
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            h_A[i*N+j] = 1;
            h_B[i*N+j] = 1;
            C_CPU[i*N+j] = 0;
        }
    }

    // Copy host array to device array
    hipMemcpy(d_A, h_A, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C_CPU, memSize, hipMemcpyHostToDevice);

    // launch kernel
    dim3 dimGrid(N/T);
    dim3 dimBlock(T);
    mtxMult<<< dimGrid, dimBlock >>>(d_A, d_B, d_C, N);

    // device to host copy
    hipMemcpy(C_GPU, d_C, memSize, hipMemcpyDeviceToHost );

    // Run program sequentially
    for (int i = 0; i < N; i++){
        for (int j = 0; j < N; j++){
            float temp = 0;
            for (int k = 0; k < N; k++){
                temp += h_A[i*N + k] * h_B[k * N+j];
            }
            C_CPU[i * N + j] = temp;
            printf("C_CPU[%d] = %f and C_GPU[%d] = %f\n", i * N + j, C_CPU[i * N + j], i * N + j, C_GPU[i * N + j]);
        }
    }

    /*
    printf("First 20 C CPU: ");
    for (int i = 0; i < 20; i++){
        printf("%f ", C_CPU[i]);
    }
    printf("\n");

    printf("First 20 C GPU: ");
    for (int i = 0; i < 20; i++){
        printf("%f ", C_GPU[i]);
    }
    printf("\n");

    printf("Verifying program correctness.... ");
    // verify the data returned to the host is correct
    for (int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            assert(C_CPU[i*N + j] == C_GPU[i*N + j]);
        }
    }
    printf("Everthing checks out!\n");*/

    // free host memory
    free(h_A);
    free(h_B);
    free(C_CPU);
    free(C_GPU);

    // free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
} // qsub hw11.sh -q UI-GPU -I ngpus=1